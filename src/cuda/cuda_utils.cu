#include "cuda/cuda_utils.h"
#include <iostream>

namespace CudaUtils {

void printDeviceInfo() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    
    if (deviceCount == 0) {
        std::cerr << "No CUDA-capable devices found!" << std::endl;
        return;
    }
    
    std::cout << "Found " << deviceCount << " CUDA device(s):" << std::endl;
    
    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        
        std::cout << "Device " << i << ": " << prop.name << std::endl;
        std::cout << "  Compute capability: " << prop.major << "." << prop.minor << std::endl;
        std::cout << "  Global memory: " << prop.totalGlobalMem / (1024 * 1024) << " MB" << std::endl;
        std::cout << "  Multiprocessors: " << prop.multiProcessorCount << std::endl;
        std::cout << "  Max threads per block: " << prop.maxThreadsPerBlock << std::endl;
    }
}

bool checkCudaCapabilities() {
    int deviceCount;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    
    if (error != hipSuccess || deviceCount == 0) {
        std::cerr << "No CUDA-capable devices found!" << std::endl;
        return false;
    }
    
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    
    // Check for minimum compute capability (3.0)
    if (prop.major < 3) {
        std::cerr << "CUDA compute capability 3.0 or higher required!" << std::endl;
        return false;
    }
    
    return true;
}

} // namespace CudaUtils